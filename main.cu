#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cassert>
#include <algorithm>
#include "SpTimer.hpp"

#include <hip/hip_runtime.h>

#define CUDA_ASSERT(X)\
{\
    hipError_t ___resCuda = (X);\
    if ( hipSuccess != ___resCuda ){\
    printf("Error: fails, %s (%s line %d)\nbCols", hipGetErrorString(___resCuda), __FILE__, __LINE__ );\
    exit(1);\
    }\
    }

template <int Idx>
struct SwitchIdx{
    static constexpr int value = Idx;

    constexpr int getValue() const{
        return value;
    }

    operator int() const{
        return Idx;
    }
};

template <int CaseVal, int... Ints, class Func>
void Switch(const int choice, Func&& f){
    if(choice == CaseVal){
        f(SwitchIdx<CaseVal>());
    }
    else {
        if constexpr (sizeof...(Ints) >= 1){
            Switch<Ints...>(choice, std::forward<Func>(f));
        }
        else{
            std::cout << "Choice does not exist in switch " << choice << std::endl;
            exit(0);
        }
    }
}

/////////////////////////////////////////////////////////////

using ValType = double;

__device__ __host__ unsigned int targetNeigh(const unsigned int thread, const unsigned int idx){
    const unsigned int pos = (thread ^ idx);
    assert(0 <= pos);
    assert(pos < 32);
    return pos;
}

__global__ void core_test(const ValType* values, ValType* results, const int nbLoops){
    const int threadIdxInWarp = threadIdx.x%32;

    ValType buffer[32];
    for(int idxVal = 0 ; idxVal < 32 ; ++idxVal){
        buffer[idxVal] = values[threadIdxInWarp*32 + idxVal];
    }


    for(int idxLoop = 0 ; idxLoop < nbLoops ; ++idxLoop){
        for(unsigned int idx = 1 ; idx < 32 ; idx *= 2){
            const unsigned int neighIdx = targetNeigh(threadIdxInWarp, idx);
            buffer[threadIdxInWarp] += __shfl_xor_sync(0xffffffff, buffer[neighIdx], idx, 32);

            const int step = idx*2;
            for(unsigned int idxCoverage = step ; idxCoverage < 32 ; idxCoverage += step){
                const unsigned int recvFor = (threadIdxInWarp + idxCoverage)%32u;
                const unsigned int sendFor = (neighIdx + idxCoverage)%32u;
                buffer[recvFor] += __shfl_xor_sync(0xffffffff, buffer[sendFor], idx, 32);
            }
        }
    }

    results[blockIdx.x*blockDim.x + threadIdx.x] = buffer[threadIdxInWarp];
}

auto test_cu_partition(const std::vector<ValType>& values,
                       const int nbGroupsTest,
                       const int nbThreadsTest,
                       const int NbLoops){
    assert(values.size() == 32*32);
    ValType* cuValues;
    CUDA_ASSERT( hipMalloc(&cuValues, 32*32 * sizeof(ValType)) );
    CUDA_ASSERT( hipMemcpy(cuValues, values.data(),
                            32*32 * sizeof(ValType),
                            hipMemcpyHostToDevice) );

    ValType* cuResults;
    CUDA_ASSERT( hipMalloc(&cuResults, nbThreadsTest*nbGroupsTest * sizeof(ValType)) );

    SpTimer timer;

    core_test<<<nbGroupsTest,nbThreadsTest>>>(cuValues, cuResults, NbLoops);
    CUDA_ASSERT(hipDeviceSynchronize());

    timer.stop();
    std::cout << "WARP = " << timer.getElapsed() << std::endl;

    std::vector<ValType> results(nbThreadsTest*nbGroupsTest);
    CUDA_ASSERT( hipMemcpy(results.data(), cuResults, nbThreadsTest*nbGroupsTest * sizeof(ValType),
                            hipMemcpyDeviceToHost) );

    CUDA_ASSERT( hipFree(cuValues) );

    return results;
}


template <int nbThreadsPerBlock>
__global__ void core_test_sm(const ValType* values, ValType* results, const int nbLoops){
    const int warpSize = 32;
    __shared__ ValType intermediateResultsAll[nbThreadsPerBlock/warpSize][warpSize][warpSize];

    const int idxWarpInBlock = (threadIdx.x/warpSize);
    const int idxThreadInWarp = (threadIdx.x%warpSize);

    const int threadIdxInWarp = threadIdx.x/32;

    ValType buffer[32];
    for(int idxVal = 0 ; idxVal < 32 ; ++idxVal){
        buffer[idxVal] = values[threadIdxInWarp*32 + idxVal];
    }


    ValType sum = 0;
    for(int idxLoop = 0 ; idxLoop < nbLoops ; ++idxLoop){
        ValType (*intermediateResults)[warpSize] = intermediateResultsAll[idxWarpInBlock];

        for(int idxVal = 0 ; idxVal < 32 ; ++idxVal){
            intermediateResults[idxVal][idxThreadInWarp] = buffer[idxVal];
        }

        for(int idxVal = 0 ; idxVal < 32 ; ++idxVal){
            sum += intermediateResults[idxThreadInWarp][idxVal];
        }
    }

    results[blockIdx.x*blockDim.x + threadIdx.x] = sum;
}


auto test_cu_partition_sm(const std::vector<ValType>& values,
                          const int nbGroupsTest,
                          const int nbThreadsTest,
                          const int NbLoops){
    assert(values.size() == 32*32);
    ValType* cuValues;
    CUDA_ASSERT( hipMalloc(&cuValues, 32*32 * sizeof(ValType)) );
    CUDA_ASSERT( hipMemcpy(cuValues, values.data(),
                            32*32 * sizeof(ValType),
                            hipMemcpyHostToDevice) );

    ValType* cuResults;
    CUDA_ASSERT( hipMalloc(&cuResults, nbThreadsTest*nbGroupsTest * sizeof(ValType)) );

    SpTimer timer;

    Switch<32, 64, 96, 128, 160, 192>(nbThreadsTest, [&](auto idx){
        core_test_sm<idx.getValue()><<<nbGroupsTest,idx.getValue()>>>(cuValues, cuResults, NbLoops);
        CUDA_ASSERT(hipDeviceSynchronize());
    });

    timer.stop();
    std::cout << "SM = " << timer.getElapsed() << std::endl;

    std::vector<ValType> results(nbThreadsTest*nbGroupsTest);
    CUDA_ASSERT( hipMemcpy(results.data(), cuResults, nbThreadsTest*nbGroupsTest * sizeof(ValType),
                            hipMemcpyDeviceToHost) );

    CUDA_ASSERT( hipFree(cuValues) );

    return results;
}

/////////////////////////////////////////////////////////////



int main(){
    std::vector<ValType> values(32*32);
    for(int idx0 = 0 ; idx0 < 32 ; ++idx0){
        for(int idx1 = 0 ; idx1 < 32 ; ++idx1){
            values[idx0*32 + idx1] = idx1+1;//(idx0 == 0 ? 1 : 0);
        }
    }

    {
        const int nbBlocksTest = 1;
        const int nbThreadsTest = 32;
        auto results = test_cu_partition(values, nbBlocksTest, nbThreadsTest, 1);
        for(int idx0 = 0 ; idx0 < 32 ; ++idx0){
            std::cout << idx0 << ") " << results[idx0] << std::endl;
        }

        auto results_sm = test_cu_partition_sm(values, nbBlocksTest, nbThreadsTest, 1);
        for(int idx0 = 0 ; idx0 < 32 ; ++idx0){
            std::cout << idx0 << ") " << results_sm[idx0] << std::endl;
        }
    }

    const int NbLoops = 10000;

    std::vector<long int> nbThreadsPossibleValues{32, 64, 96, 128, 160, 192};
    for(long int nbBlocksTest = 16 ; nbBlocksTest < 1024/*2147483647*/ ; nbBlocksTest *= 2){
        for(long int nbThreadsTest : nbThreadsPossibleValues){
            std::cout << " nbBlocksTest " << nbBlocksTest
                      << " nbThreadsTest " << nbThreadsTest << std::endl;

            test_cu_partition(values, nbBlocksTest, nbThreadsTest, NbLoops);

            test_cu_partition_sm(values, nbBlocksTest, nbThreadsTest, NbLoops);
        }
    }

	return 0;
}
